
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */
 
 
__global__ void GPU_loop() {

    int block = blockIdx.x;
    int thread = threadIdx.x;
    
    printf("This is core number %d\n", block*blockDim.x + thread);

}

void loop(int N)
{
  for (int i = 0; i < N; ++i)
  {
    printf("This is iteration number %d\n", i);
  }
}

int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, be sure to use more than 1 block in
   * the execution configuration.
   */

  int N = 10;
  //loop(N);
  
  int numBlocksToUse = 2;
  
  GPU_loop <<<numBlocksToUse,N/numBlocksToUse>>>();
  hipDeviceSynchronize();
}
