
#include <hip/hip_runtime.h>
#include <stdio.h>

void initWith(float num, float *a, int N)
{
  for(int i = 0; i < N; ++i)
  {
    a[i] = num;
  }
}

__global__ void initWith_GPU(float num, float *a, int N) {
    
    int gridIndex = threadIdx.x + blockIdx.x*blockDim.x;
    int strideSize = gridDim.x*blockDim.x;
    
    for (int i = gridIndex; i < N; i = i + strideSize) {
        a[i] = num;
    }
}

void addVectorsInto(float *result, float *a, float *b, int N)
{
  for(int i = 0; i < N; ++i)
  {
    result[i] = a[i] + b[i];
  }
}

__global__ void addVectorsInto_GPU(float *result, float *a, float *b, int N) {
    
    int gridIndex = threadIdx.x + blockIdx.x*blockDim.x;
    int strideSize = gridDim.x*blockDim.x;
     
    for (int i = gridIndex; i < N; i = i + strideSize) {
        result[i] = a[i] + b[i];
    }
    
}

void checkElementsAre(float target, float *array, int N)
{
  for(int i = 0; i < N; i++)
  {
    if(array[i] != target)
    {
      printf("FAIL: array[%d] - %0.0f does not equal %0.0f\n", i, array[i], target);
      exit(1);
    }
  }
  printf("SUCCESS! All values added correctly.\n");
}

int main()
{
  const int N = 2<<20;
  size_t size = N * sizeof(float);

  float *a;
  float *b;
  float *c;

  //a = (float *)malloc(size);
  //b = (float *)malloc(size);
  //c = (float *)malloc(size);
  
  hipMallocManaged(&a, size);
  hipMallocManaged(&b, size);
  hipMallocManaged(&c, size);
    
  int numThreads = 1024;
  int numBlocks = 32;

  // initWith(3, a, N);
  // initWith(4, b, N);
  // initWith(0, c, N);
  
  initWith_GPU<<<numBlocks,numThreads>>>(3, a, N);
  initWith_GPU<<<numBlocks,numThreads>>>(4, b, N);
  initWith_GPU<<<numBlocks,numThreads>>>(0, c, N);
    
  hipDeviceSynchronize();

  // addVectorsInto(c, a, b, N);
  addVectorsInto_GPU<<<numBlocks,numThreads>>>(c, a, b, N);
    
  hipDeviceSynchronize();

  checkElementsAre(7, c, N);

  // free(a);
  // free(b);
  // free(c);
    
  hipFree(a);
  hipFree(b);
  hipFree(c);
}
